/* CUDA device manager by Dan Liew & Alex Allen */
#include <stdio.h>
#include "devicemanager.h"

bool deviceErrorHandle(hipError_t error)
{
	switch(error)
	{
		case hipSuccess:
			//fprintf(stderr,"CUDA Success.");
			return true;

		
		default:
			fprintf(stderr,"CUDA Error: CUDA %s.",hipGetErrorString(error));
			return false;
	}

}
