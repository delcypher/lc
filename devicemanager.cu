#include "hip/hip_runtime.h"
/* CUDA device manager by Dan Liew & Alex Allen */
#include <stdio.h>
#include "devicemanager.h"

bool deviceErrorHandle(hipError_t error)
{
	switch(error)
	{
		case hipSuccess:
			//fprintf(stderr,"CUDA Success.\n");
			return true;

		
		default:
			fprintf(stderr,"CUDA Error: %s.\n",hipGetErrorString(error));
			return false;
	}

}

int pickGPU(int maj, int min)
{
        int dev;
	int deviceCount;

        // Create hipDeviceProp_t with the specifications we want
        hipDeviceProp_t prop;
        memset(&prop, 0, sizeof(hipDeviceProp_t));
        prop.major = maj;
        prop.minor = min;

	//get the number of CUDA devices.	
	deviceErrorHandle( hipGetDeviceCount(&deviceCount) );

	if(deviceCount==0)
	{
		fprintf(stderr,"Error: No CUDA devices available.\n");
		exit(1);
	}

        // Use built in functions to pick a device with those specs
        deviceErrorHandle( hipChooseDevice(&dev, &prop) );
        deviceErrorHandle( hipSetDevice(dev) );

        return dev;
}

