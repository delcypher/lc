#include "hip/hip_runtime.h"
/* Main section of code where the LatticeObject is setup and processed
*  By Alex Allen & Daniel Liew (2010)
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include "randgen.h"
#include "differentiate.h"
#include "lattice.h"
#include "dev_lattice.cuh"
#include "dev_differentiate.cuh"
#include "devicemanager.h"

/* CUDA does not support external (i.e. to other objects used in iterative compilation) function calls.
*  So we must include the implementations of the device functions in the same file (and hence object) as
*  our kernel function(s). This slows down compilation but at least it works!
*/
#include "dev_lattice.cu"
#include "dev_differentiate.cu"

using namespace std;

/* Include the nanoparticle header files you wish you use here.
*  Make sure the nanoparticle is listed in the OBJECTS variable
*  in the make file too!
*/
#include "nanoparticles/circle.h"

const int threadDim = 8;

__global__ void kernel(LatticeObject *baconlatticetomato, double *blockEnergies);

int main()
{
	LatticeConfig configuration;
	FILE* fout = fopen("dump.txt", "w");

	cout << "# Setting lattice config parameters" << endl;	
	//setup lattice parameters
	configuration.width = threadDim*10;
	configuration.height= threadDim*10;

	//set initial director alignment
	configuration.initialState = LatticeConfig::RANDOM;

	//set boundary conditions
	configuration.topBoundary = LatticeConfig::BOUNDARY_PARALLEL;
	configuration.bottomBoundary = LatticeConfig::BOUNDARY_PERPENDICULAR;
	configuration.leftBoundary = LatticeConfig::BOUNDARY_PERIODIC;
	configuration.rightBoundary = LatticeConfig::BOUNDARY_PERIODIC;

	//set lattice beta value
	configuration.beta = 3.5;

	//pick a GPU to use
	int deviceSelected = pickGPU(1,3);
	if(deviceSelected==-1)
	{
		exit(1);
	}

	printf("# Selecting CUDA device:%d \n",deviceSelected);

	//create lattice object, with (configuration, dump precision)
	Lattice nSystem = Lattice(configuration,10);

	cout << "# Creating nanoparticle" << endl; 

	//create circular nanoparticle (x,y,radius, boundary)
	CircularNanoparticle particle1 = CircularNanoparticle(10,10,5,CircularNanoparticle::PARALLEL);
	
	cout << "# Adding nanoparticle" << endl;

	//add nanoparticle to lattice
	nSystem.add(&particle1);

	cout << "# Initialise lattice on device" << endl;

	//Initialise the lattice on the device
	nSystem.initialiseCuda();
	
	//Dump the current state of the lattice to standard output.
	//nSystem.nDump(Lattice::BOUNDARY,stdout);
	nSystem.indexedNDump(fout);

        //Alex's wizardry
        int xblocks = configuration.width/threadDim, yblocks = configuration.height/threadDim;
	int arraySize = xblocks*threadDim * yblocks*threadDim;
        dim3 blocks(xblocks, yblocks);
        dim3 threads(threadDim, threadDim);

	cout << "# Create variables and allocate device memory" << endl;

        double totalEnergy=0, blockEnergies[arraySize], *dev_blockEnergies;
        deviceErrorHandle(hipMalloc((void**) &dev_blockEnergies, arraySize*sizeof(double)));

	cout << "# Run kernel" << endl;
        kernel<<<blocks, threads>>>(nSystem.devLatticeObject, dev_blockEnergies);
   
	cout << "# Copy energy from device to host" << endl;
	hipMemcpy(blockEnergies, dev_blockEnergies, arraySize*sizeof(double), hipMemcpyDeviceToHost);
	
	cout << "# Copy nSystem from device to host" << endl; 
	nSystem.copyDeviceToHost();

	cout << "# Sum block energies" << endl;
        int i;
        for(i=0; i<arraySize; i++)
        {
                totalEnergy+=blockEnergies[i];
        }

	//Dump the current state of the lattice to standard output.
	//nSystem.nDump(Lattice::EVERYTHING,stdout);

	double energy = nSystem.calculateTotalEnergy();
	cout << "# Energy calculated on CPU: " << energy << endl;
	cout << "# Energy calculated on GPU: " << totalEnergy << endl;
/*
	cout << "# Block energies were: ";

	for(i=0; i<arraySize; i++)
	{
		cout << blockEnergies[i] << " ";
	}
	cout << endl;
*/
	fclose(fout);

	return 0;
}

__global__ void kernel(LatticeObject *baconlatticetomato, double *blockEnergies)
{
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;
        int threadID = x + y * blockDim.x * gridDim.x;

        blockEnergies[threadID] = latticeCalculateEnergyOfCell(baconlatticetomato, x,y);
}
